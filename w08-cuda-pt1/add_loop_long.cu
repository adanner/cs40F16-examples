#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2010 NVIDIA Corporation.  All rights reserved.
 *
 * NVIDIA Corporation and its licensors retain all intellectual property and 
 * proprietary rights in and to this software and related documentation. 
 * Any use, reproduction, disclosure, or distribution of this software 
 * and related documentation without an express license agreement from
 * NVIDIA Corporation is strictly prohibited.
 *
 * Please refer to the applicable NVIDIA end user license agreement (EULA) 
 * associated with this source code for terms and conditions that govern 
 * your use of this NVIDIA software.
 * 
 */


#include "book.h"

#define N   (32 * 1024 * 1024)

__global__ void add( int *a, int *b, int *c ) {
    int tid = blockIdx.x;
    while (tid < N) {
        c[tid] = a[tid] + b[tid];
        tid += gridDim.x;
    }
}

int main( void ) {
    int *a, *b, *c;
    int *dev_a, *dev_b, *dev_c;
    hipEvent_t start,stop;
float elapsed_ms; //elapsed time in milliseconds

    // allocate the memory on the CPU
    a = (int*)malloc( N * sizeof(int) );
    b = (int*)malloc( N * sizeof(int) );
    c = (int*)malloc( N * sizeof(int) );

    // allocate the memory on the GPU
    HANDLE_ERROR( hipMalloc( (void**)&dev_a, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_b, N * sizeof(int) ) );
    HANDLE_ERROR( hipMalloc( (void**)&dev_c, N * sizeof(int) ) );

    // fill the arrays 'a' and 'b' on the CPU
    for (int i=0; i<N; i++) {
        a[i] = i;
        b[i] = 2 * i;
    }

    // copy the arrays 'a' and 'b' to the GPU
    HANDLE_ERROR( hipMemcpy( dev_a, a, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    HANDLE_ERROR( hipMemcpy( dev_b, b, N * sizeof(int),
                              hipMemcpyHostToDevice ) );
    
    HANDLE_ERROR( hipEventCreate(&start) );
    HANDLE_ERROR( hipEventCreate(&stop) );
    HANDLE_ERROR( hipEventRecord(start,0) );

    add<<<128,1>>>( dev_a, dev_b, dev_c );
    HANDLE_ERROR( hipEventRecord(stop,0) );
    HANDLE_ERROR( hipEventSynchronize(stop) );
    HANDLE_ERROR( hipEventElapsedTime(&elapsed_ms, start, stop) );
    printf("Time to run kernel: %3.1f ms\n", elapsed_ms);

    HANDLE_ERROR( hipEventDestroy(start) );
    HANDLE_ERROR( hipEventDestroy(stop) );

    // copy the array 'c' back from the GPU to the CPU
    HANDLE_ERROR( hipMemcpy( c, dev_c, N * sizeof(int),
                              hipMemcpyDeviceToHost ) );

    // verify that the GPU did the work we requested
    bool success = true;
    for (int i=0; i<N; i++) {
        if ((a[i] + b[i]) != c[i]) {
            printf( "Error:  %d + %d != %d\n", a[i], b[i], c[i] );
            success = false;
        }
    }
    if (success)    printf( "We did it!\n" );

    // free the memory we allocated on the GPU
    HANDLE_ERROR( hipFree( dev_a ) );
    HANDLE_ERROR( hipFree( dev_b ) );
    HANDLE_ERROR( hipFree( dev_c ) );

    // free the memory we allocated on the CPU
    free( a );
    free( b );
    free( c );

    return 0;
}

