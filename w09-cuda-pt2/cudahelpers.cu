#include "hip/hip_runtime.h"
#include "cudahelpers.h"
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <cstdio>

#define DIM 1000
__device__ int julia( int x, int y, float re, float im);
__global__ void kernel( unsigned char *ptr, float re, float im);

MyCUDAWrapper::MyCUDAWrapper():m_pbo_CUDA(NULL){};

void MyCUDAWrapper::init(){
    cudaGLSetGLDevice(0);
}

void MyCUDAWrapper::connect(GLuint buffID){
    if(m_pbo_CUDA){ disconnect(); }
    hipGraphicsGLRegisterBuffer(&m_pbo_CUDA,buffID,hipGraphicsRegisterFlagsNone);
}

void MyCUDAWrapper::disconnect(){
    if(m_pbo_CUDA){
        hipGraphicsUnregisterResource(m_pbo_CUDA);
    }
}

void MyCUDAWrapper::run(float a, float b){

    unsigned char* dev_pixBuffer;
    size_t numBytes;
    dim3    grid(DIM,DIM);

    // Map buffer object for writing from CUDA
    hipGraphicsMapResources(1, &m_pbo_CUDA);
    hipGraphicsResourceGetMappedPointer((void**)&dev_pixBuffer,
                                         &numBytes,
                                         m_pbo_CUDA);

    kernel<<<grid,1>>>( dev_pixBuffer, a, b);
    hipDeviceSynchronize(); //Make sure kernel is done
    //Return PBO to OpenGL control.
    hipGraphicsUnmapResources(1, &m_pbo_CUDA);
}


struct hipComplex {
    float   r;
    float   i;
    __device__ hipComplex( float a, float b ) : r(a), i(b)  {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ hipComplex operator*(const hipComplex& a) {
        return hipComplex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ hipComplex operator+(const hipComplex& a) {
        return hipComplex(r+a.r, i+a.i);
    }
};

__device__ int julia( int x, int y, float re, float im) {
    const float scale = 1.5;
    float jx = scale * (float)(DIM/2 - x)/(DIM/2);
    float jy = scale * (float)(DIM/2 - y)/(DIM/2);

    hipComplex c(re, im);
    hipComplex a(jx, jy);

    int i = 0;
    for (i=0; i<200; i++) {
        a = a * a + c;
        if (a.magnitude2() > 1000)
            return 0;
    }

    return 1;
}

__global__ void kernel( unsigned char *ptr, float re, float im ) {
    // map from threadIdx/BlockIdx to pixel position
    int x = blockIdx.x;
    int y = blockIdx.y;
    int offset = x + y * gridDim.x;

    // now calculate the value at that position
    int juliaValue = julia( x, y, re, im);
    ptr[offset*4 + 0] = 255 * juliaValue;
    ptr[offset*4 + 1] = 0;
    ptr[offset*4 + 2] = 64*(1-juliaValue);
    ptr[offset*4 + 3] = 255;
}




